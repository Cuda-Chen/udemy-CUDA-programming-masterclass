#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void print_threadIds_blockIds_gridDim()
{
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d,\
    blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d,\
    gridDim.x: %d, gridDim.y: %d, gridDim.z: %d \n",
    threadIdx.x, threadIdx.y, threadIdx.z,
    blockIdx.x, blockIdx.y, blockIdx.z,
    gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    int nx, ny, nz;
    nx = 4;
    ny = 4;
    nz = 4;

    dim3 block(2, 2, 2);
    dim3 grid(nx / block.x, ny / block.y, nz / block.z);

    print_threadIds_blockIds_gridDim<<<grid, block>>>();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}
